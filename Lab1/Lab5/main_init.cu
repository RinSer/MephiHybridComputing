#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <ctype.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <errno.h>
#include <string.h>
#include <vector>
#include <time.h>
// CUDA runtime
#include <hip/hip_runtime.h>
// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#define MAX_SIZE 12

int process_stream(int input_port, int output_port, int devId);

int main(int argc, char* argv[])
{
    if (argc < 3)
    {
        printf("Need input and output streams ports as parameters!\n");
        return -1;
    }

    int input_port = atoi(argv[1]);
    int output_port = atoi(argv[2]);

    int devId = findCudaDevice(argc, (const char **)argv);

    for (;;) // forever and ever
    {
        printf("I am waiting for a Matrix at port %d\n", input_port);
        process_stream(input_port, output_port, devId);
    }

    return 0;
}

double getMilliseconds() {
    return 1000.0 * clock() / CLOCKS_PER_SEC;
}

int* listen_to_port(int port)
{
    int sfd, connection;

    int* opts = new int[1]{ 1 };

    if ((sfd = socket(AF_INET, SOCK_STREAM, 0)) == 0
        || setsockopt(sfd, SOL_SOCKET, SO_REUSEADDR | SO_REUSEPORT, opts, sizeof(int)))
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    struct sockaddr_in address;
    int addrlen = sizeof(address);
    address.sin_family = AF_INET;
    address.sin_addr.s_addr = INADDR_ANY;
    address.sin_port = htons(port);

    if (bind(sfd, (struct sockaddr*)&address,
        sizeof(address)) < 0 || listen(sfd, 3) < 0
        || (connection = accept(sfd, (struct sockaddr*)&address, (socklen_t*)&addrlen)) < 0)
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    return new int[2]{ connection, sfd };
}

void close_connection(int connection, int sfd) 
{
    shutdown(connection, SHUT_RDWR);
    close(connection);
    shutdown(sfd, SHUT_RDWR);
    close(sfd);
}

void send_stream_to_port(int port, char* stream)
{
    int sock = 0;
    if ((sock = socket(AF_INET, SOCK_STREAM, 0)) < 0)
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    struct sockaddr_in serv_addr;
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_addr.s_addr = INADDR_ANY;
    serv_addr.sin_port = htons(port);

    if (connect(sock, (struct sockaddr*)&serv_addr, sizeof(serv_addr)) < 0)
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    send(sock, stream, strlen(stream), 0);

    shutdown(sock, SHUT_WR);
    close(sock);
}

int get_stream_matrix(int connection, std::vector<std::vector<float>> &matrix)
{
    int buffer_size = 1;
    char* buffer = new char[buffer_size];
    
    std::vector<char> number;
    std::vector<float> line;
    int stream_size = 0;
    while (read(connection, buffer, buffer_size) > 0)
    {
        stream_size++;
        int num_bytes = 1;

        while (buffer[0] != '\n' && num_bytes > 0)
        {
            if (buffer[0] == ' ')
            {
                line.push_back(atof(&number[0]));
                number.clear();
            }
            else
                number.push_back(buffer[0]);

            num_bytes = read(connection, buffer, buffer_size);
            stream_size++;
        }
        line.push_back(atof(&number[0]));
        number.clear();

        matrix.push_back(line);

        line.clear();
    }

    return stream_size;
}

void convertVectorToFlatArray(std::vector<std::vector<float>> matrix, float *flat)
{
    int msize = matrix.size();
    for (int i = 0; i < msize; i++)
    { 
        for (int j = 0; j < msize; j++)
        {
            flat[j + (i * msize)] = matrix[i][j];
        }
    }
 }

__global__ void get_avg_vector(float *matrix, float *result, int n)
{
    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nsqrd = n*n;

    if (row_idx < nsqrd)
    {
        for (int i = row_idx; i < nsqrd; i += blockDim.x * gridDim.x)
            atomicAdd(&result[i / n], matrix[i] / n);
    }
    
}

int process_stream(int input_port, int output_port, int devId)
{
    int* connection = listen_to_port(input_port);

    std::vector<std::vector<float>> matrix;

    int stream_size = get_stream_matrix(connection[0], matrix);

    close_connection(connection[0], connection[1]);

    int matrix_size = matrix.size();
    int row_size = matrix_size * sizeof(float);
    int flat_size = matrix_size * matrix_size * sizeof(float);

    float *a_matrix = (float*)malloc(flat_size);
    convertVectorToFlatArray(matrix, a_matrix);

    float *d_matrix;
    float *d_result;

    hipMalloc(&d_matrix, flat_size);
    hipMemcpy(d_matrix, a_matrix, flat_size, hipMemcpyHostToDevice);
    
    hipMalloc(&d_result, row_size);
    hipMemset(d_result, 0, row_size);

    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, devId);
    printf("Executed on device \"%s\"\n", props.name);

    double start = getMilliseconds();

    get_avg_vector<<<numSMs, 256>>>(d_matrix, d_result, matrix_size);

    double end = getMilliseconds();
    double execution_time_in_seconds = (double)(end - start);

    float *float_result = (float*)malloc(row_size);
    hipMemcpy(float_result, d_result, row_size, hipMemcpyDeviceToHost);

    std::vector<char> result;

    for (int i = 0; i < matrix_size; i++) 
    {
        char* avg_str = new char[MAX_SIZE];
        sprintf(avg_str, "%.8f ", float_result[i]);
        result.insert(result.end(), avg_str, avg_str + strlen(avg_str));
    }
    result.push_back('\n');

    char* buffer = new char[256];
    sprintf(buffer, "%d bytes in %.9f milliseconds\n", stream_size, execution_time_in_seconds);

    result.insert(result.end(), buffer, buffer + strlen(buffer));
    result.push_back('\0');

    send_stream_to_port(output_port, &result[0]);

    hipFree(d_result);
    hipFree(d_matrix);
    free(float_result);
    free(a_matrix);

    return 0;
}