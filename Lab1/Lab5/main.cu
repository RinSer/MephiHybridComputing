#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <ctype.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <string.h>
#include <time.h>
#include <vector>
#include <math.h>
#include <mpi.h>
#define WORD_SIZE 11
#define MAX_SIZE 12

int* listen_to_port(int port)
{
    int sfd, connection;

    int* opts = new int[1]{ 1 };

    if ((sfd = socket(AF_INET, SOCK_STREAM, 0)) == 0
        || setsockopt(sfd, SOL_SOCKET, SO_REUSEADDR | SO_REUSEPORT, opts, sizeof(int)))
        exit(EXIT_FAILURE);

    struct sockaddr_in address;
    int addrlen = sizeof(address);
    address.sin_family = AF_INET;
    address.sin_addr.s_addr = INADDR_ANY;
    address.sin_port = htons(port);

    if (bind(sfd, (struct sockaddr*)&address,
        sizeof(address)) < 0 || listen(sfd, 3) < 0
        || (connection = accept(sfd, (struct sockaddr*)&address, (socklen_t*)&addrlen)) < 0)
        exit(EXIT_FAILURE);

    return new int[2]{ connection, sfd };
}

void close_connection(int connection, int sfd)
{
    shutdown(connection, SHUT_RDWR);
    close(connection);
    shutdown(sfd, SHUT_RDWR);
    close(sfd);
}

void send_stream_to_port(int port, char* stream)
{
    int sock = 0;
    if ((sock = socket(AF_INET, SOCK_STREAM, 0)) < 0)
        exit(EXIT_FAILURE);

    struct sockaddr_in serv_addr;
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_addr.s_addr = INADDR_ANY;
    serv_addr.sin_port = htons(port);

    if (connect(sock, (struct sockaddr*)&serv_addr, sizeof(serv_addr)) < 0)
        exit(EXIT_FAILURE);

    send(sock, stream, strlen(stream), 0);

    shutdown(sock, SHUT_WR);
    close(sock);
}

float get_line_avg(std::vector<float> line)
{
    int count = line.size();
    float sum = 0;

    for (int i = 0; i < count; i++)
        sum += line[i] / count;

    return sum;
}

std::vector<char> get_avg_vector(std::vector<float> matrix, int row_size, int num_rows)
{
    std::vector<float> result;
    result.reserve(num_rows);

    for (int i = 0; i < num_rows; i++)
    {
        double avg = get_line_avg(std::vector<float>(matrix.begin() + (i * row_size), matrix.begin() + (i * row_size) + row_size));
        result[i] = avg;
    }

    std::vector<char> char_result;

    for (int i = 0; i < num_rows; i++)
    {
        char* avg_str = new char[MAX_SIZE];
        sprintf(avg_str, "%.8f ", result[i]);
        char_result.insert(char_result.end(), avg_str, avg_str + strlen(avg_str));
    }

    return char_result;
}

int get_stream_matrix(int connection, std::vector<std::vector<float>>& matrix)
{
    int buffer_size = 1;
    char* buffer = new char[buffer_size];

    std::vector<char> number;
    std::vector<float> line;
    int stream_size = 0;
    while (read(connection, buffer, buffer_size) > 0)
    {
        stream_size++;
        int num_bytes = 1;

        while (buffer[0] != '\n' && num_bytes > 0)
        {
            if (buffer[0] == ' ')
            {
                line.push_back(atof(&number[0]));
                number.clear();
            }
            else
                number.push_back(buffer[0]);

            num_bytes = read(connection, buffer, buffer_size);
            stream_size++;
        }
        line.push_back(atof(&number[0]));
        number.clear();

        matrix.push_back(line);

        line.clear();
    }

    return stream_size;
}

int main(int argc, char* argv[])
{
    if (argc < 3)
    {
        printf("Need input and output streams ports as parameters!");
        return -1;
    }

    int input_port = atoi(argv[1]);
    int output_port = atoi(argv[2]);

    MPI_Init(&argc, &argv);

    for (;;) // forever and ever
    {
        int rank, numtasks, stream_size, num_elements, row_size;

        std::vector<std::vector<float>> matrix;
        std::vector<float> flat_matrix;
        std::vector<float> matrix_row;

        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &numtasks);

        int* sendcounts = new int[numtasks];
        int* displs = new int[numtasks];

        if (rank == 0) // only the first process communicates with external
        {
            int* connection = listen_to_port(input_port);

            stream_size = get_stream_matrix(connection[0], matrix);

            close_connection(connection[0], connection[1]);

            int total_size = matrix.size() * matrix.size();
            
            for (int i = 0; i < matrix.size(); i++)
                for (int j = 0; j < matrix[i].size(); j++)
                    flat_matrix.push_back(matrix[i][j]);

            row_size = (int)matrix.size();
            num_elements = (int)floor(total_size / numtasks);
            num_elements -= num_elements % row_size;

            // calculate send counts and displacements
            int sum = 0;
            for (int i = 0; i < numtasks; i++) {
                sendcounts[i] = num_elements;

                if (i == numtasks - 1)
                    sendcounts[i] = total_size - sum;

                displs[i] = sum;
                sum += sendcounts[i];
            }
        }

        MPI_Barrier(MPI_COMM_WORLD);

        MPI_Bcast(&num_elements, 1, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Bcast(&row_size, 1, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Bcast(sendcounts, numtasks, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Bcast(displs, numtasks, MPI_INT, 0, MPI_COMM_WORLD);

        int num_rows = sendcounts[rank] / row_size;

        std::vector<float> partial_matrix;
        partial_matrix.resize(row_size * row_size);

        double start;
        if (rank == 0) start = MPI_Wtime(); // only the first process controls the timing

        MPI_Scatterv(flat_matrix.data(), sendcounts, displs, MPI_FLOAT, partial_matrix.data(), sendcounts[numtasks-1], MPI_FLOAT, 0, MPI_COMM_WORLD);

        // TODO: move to CUDA kernel !!!
        std::vector<char> partial_result = get_avg_vector(partial_matrix, row_size, num_rows);

        std::vector<char> parallel_result;
        if (rank == 0) parallel_result.reserve(WORD_SIZE * row_size);

        int* recvcounts = new int[numtasks];
        int* rdispls = new int[numtasks];
        int sum = 0;
        int char_size = num_elements / row_size * WORD_SIZE;
        for (int i = 0; i < numtasks; i++) {
            recvcounts[i] = char_size;

            if (i == numtasks - 1)
                recvcounts[i] = (row_size * WORD_SIZE) - sum;

            rdispls[i] = sum;
            sum += recvcounts[i];
        }

        MPI_Gatherv(partial_result.data(), (int)partial_result.size(), MPI_CHAR, parallel_result.data(), recvcounts, rdispls, MPI_CHAR, 0, MPI_COMM_WORLD);

        if (rank == 0) // only the first process communicates with external
        {
            char* result = new char[sum + 2];
            snprintf(result, sum + 1, "%s", &parallel_result[0]);
            
            double end = MPI_Wtime();
            double execution_time_in_seconds = (double)(end - start) * 1000;

            char* buffer = new char[sum + 3 + 256];
            snprintf(buffer, sum + 2 + 256, "%s\n%d bytes in %.9f milliseconds\n", result, stream_size, execution_time_in_seconds);
            printf("%s\n", buffer);

            send_stream_to_port(output_port, buffer);
        }
    }

    MPI_Finalize();

    return 0;
}