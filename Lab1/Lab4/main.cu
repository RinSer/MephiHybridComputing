#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <ctype.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <arpa/inet.h>
#include <errno.h>
#include <string.h>
#include <vector>
#include <time.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#define MAX_SIZE 12

int process_stream(int input_port, int output_port);

int main(int argc, char* argv[])
{
    if (argc < 3)
    {
        printf("Need input and output streams ports as parameters!\n");
        return -1;
    }

    int input_port = atoi(argv[1]);
    int output_port = atoi(argv[2]);

    for (;;) // forever and ever
    {
        printf("I am waiting for a Matrix at port %d\n", input_port);
        process_stream(input_port, output_port);
    }

    return 0;
}

double getMilliseconds() {
    return 1000.0 * clock() / CLOCKS_PER_SEC;
}

int* listen_to_port(int port)
{
    int sfd, connection;

    int* opts = new int[1]{ 1 };

    if ((sfd = socket(AF_INET, SOCK_STREAM, 0)) == 0
        || setsockopt(sfd, SOL_SOCKET, SO_REUSEADDR | SO_REUSEPORT, opts, sizeof(int)))
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    struct sockaddr_in address;
    int addrlen = sizeof(address);
    address.sin_family = AF_INET;
    address.sin_addr.s_addr = INADDR_ANY;
    address.sin_port = htons(port);

    if (bind(sfd, (struct sockaddr*)&address,
        sizeof(address)) < 0 || listen(sfd, 3) < 0
        || (connection = accept(sfd, (struct sockaddr*)&address, (socklen_t*)&addrlen)) < 0)
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    return new int[2]{ connection, sfd };
}

void close_connection(int connection, int sfd) 
{
    shutdown(connection, SHUT_RDWR);
    close(connection);
    shutdown(sfd, SHUT_RDWR);
    close(sfd);
}

void send_stream_to_port(int port, char* stream)
{
    int sock = 0;
    if ((sock = socket(AF_INET, SOCK_STREAM, 0)) < 0)
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    struct sockaddr_in serv_addr;
    serv_addr.sin_family = AF_INET;
    serv_addr.sin_addr.s_addr = INADDR_ANY;
    serv_addr.sin_port = htons(port);

    if (connect(sock, (struct sockaddr*)&serv_addr, sizeof(serv_addr)) < 0)
    {
        printf("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }

    send(sock, stream, strlen(stream), 0);

    shutdown(sock, SHUT_WR);
    close(sock);
}

int get_stream_matrix(int connection, std::vector<std::vector<float>> &matrix)
{
    int buffer_size = 1;
    char* buffer = new char[buffer_size];
    
    std::vector<char> number;
    std::vector<float> line;
    int stream_size = 0;
    while (read(connection, buffer, buffer_size) > 0)
    {
        stream_size++;
        int num_bytes = 1;

        while (buffer[0] != '\n' && num_bytes > 0)
        {
            if (buffer[0] == ' ')
            {
                line.push_back(atof(&number[0]));
                number.clear();
            }
            else
                number.push_back(buffer[0]);

            num_bytes = read(connection, buffer, buffer_size);
            stream_size++;
        }
        line.push_back(atof(&number[0]));
        number.clear();

        matrix.push_back(line);

        line.clear();
    }

    return stream_size;
}

void convertVectorToFlatArray(std::vector<std::vector<float>> matrix, float *flat)
{
    int msize = matrix.size();
    for (int i = 0; i < msize; i++)
    { 
        for (int j = 0; j < msize; j++)
        {
            flat[j + (i * msize)] = matrix[i][j];
        }
    }
 }

__global__ void get_avg_vector(float *matrix, float *result)
{
    int row_idx = blockIdx.x;
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    atomicAdd(&result[row_idx], matrix[idx] / blockDim.x);
}

int process_stream(int input_port, int output_port)
{
    int* connection = listen_to_port(input_port);

    std::vector<std::vector<float>> matrix;

    int stream_size = get_stream_matrix(connection[0], matrix);

    close_connection(connection[0], connection[1]);

    float *a_matrix = (float*)malloc(matrix.size() * matrix.size() * sizeof(float));
    convertVectorToFlatArray(matrix, a_matrix);

    int matrix_size = matrix.size();

    printf("%d\n", matrix_size);

    float *d_matrix;
    float *d_result;

    int row_size = matrix_size * sizeof(float);
    int flat_size = row_size * row_size;

    hipMalloc((void**)&d_matrix, flat_size);
    hipMemcpy(d_matrix, a_matrix, flat_size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_result, row_size);
    hipMemset(d_result, 0, row_size);

    double start = getMilliseconds();

    printf("%f\n", start);

    get_avg_vector<<<matrix_size, matrix_size>>>(d_matrix, d_result);
    hipDeviceSynchronize();

    double end = getMilliseconds();
    double execution_time_in_seconds = (double)(end - start);

    float *float_result = (float*)malloc(row_size);
    hipMemcpy(float_result, d_result, row_size, hipMemcpyDeviceToHost);

    std::vector<char> result;

    for (int i = 0; i < matrix_size; i++) 
    {
        char* avg_str = new char[MAX_SIZE];
        sprintf(avg_str, "%.8f ", float_result[i]);
        result.insert(result.end(), avg_str, avg_str + strlen(avg_str));
    }
    result.push_back('\n');

    char* buffer = new char[256];
    sprintf(buffer, "%d bytes in %.9f milliseconds\n", stream_size, execution_time_in_seconds);

    result.insert(result.end(), buffer, buffer + strlen(buffer));
    result.push_back('\0');

    printf("%s\n", &result[0]);

    send_stream_to_port(output_port, &result[0]);

    hipFree(d_result);
    hipFree(d_matrix);
    free(float_result);
    free(a_matrix);

    return 0;
}